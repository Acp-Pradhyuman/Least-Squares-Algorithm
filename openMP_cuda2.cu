#include <iostream>
#include <omp.h>
#include <cmath>
#include <hip/hip_runtime.h>
using namespace std;

const double LsEpsilon = 1.0e-12;

// CUDA kernel for calculating sums for least squares
__global__ void calcLeastSquaresKernel(const double* x, const double* y, int n, 
    double* sumX, double* sumY, double* sumXY, double* sumXX) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        atomicAdd(sumX, x[idx]);
        atomicAdd(sumY, y[idx]);
        atomicAdd(sumXX, x[idx] * x[idx]);
        atomicAdd(sumXY, x[idx] * y[idx]);
    }
}

// y = mx + b (Least Squares Calculation)
bool calcLeastSquaresCUDA(const double* x, const double* y, int n, 
    double* m, double* b) {
    if (n <= 0) {
        return false;
    }

    double *d_x, *d_y, *d_sumX, *d_sumY, *d_sumXY, *d_sumXX;
    double h_sumX = 0, h_sumY = 0, h_sumXY = 0, h_sumXX = 0;
    
    hipMalloc(&d_x, n * sizeof(double));
    hipMalloc(&d_y, n * sizeof(double));
    hipMalloc(&d_sumX, sizeof(double));
    hipMalloc(&d_sumY, sizeof(double));
    hipMalloc(&d_sumXY, sizeof(double));
    hipMalloc(&d_sumXX, sizeof(double));

    hipMemcpy(d_x, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_sumX, &h_sumX, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_sumY, &h_sumY, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_sumXY, &h_sumXY, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_sumXX, &h_sumXX, sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Start time before launching the kernel
    // double start_time = omp_get_wtime();
    // End time
    double start_time = static_cast<double>(clock()) / CLOCKS_PER_SEC;

    // Launch kernel to calculate sums in parallel on the GPU
    calcLeastSquaresKernel<<<gridSize, blockSize>>>(d_x, d_y, n, 
        d_sumX, d_sumY, d_sumXY, d_sumXX);

    // Wait for GPU to finish (ensure synchronization before measuring time)
    hipDeviceSynchronize();

    // End time after synchronization
    // double end_time = omp_get_wtime();
    // End time
    double end_time = static_cast<double>(clock()) / CLOCKS_PER_SEC;

    double time_taken = end_time - start_time;
    printf("Time: %f seconds\n", time_taken);

    // Copy the results back to host
    hipMemcpy(&h_sumX, d_sumX, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sumY, d_sumY, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sumXY, d_sumXY, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sumXX, d_sumXX, sizeof(double), hipMemcpyDeviceToHost);

    double denom = n * h_sumXX - h_sumX * h_sumX;
    if (LsEpsilon >= fabs(denom)) {
        return false;
    }

    *m = (n * h_sumXY - h_sumX * h_sumY) / denom;
    *b = (h_sumXX * h_sumY - h_sumX * h_sumXY) / denom;

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_sumX);
    hipFree(d_sumY);
    hipFree(d_sumXY);
    hipFree(d_sumXX);

    return true;
}

const int n = 5000 * 5000;
double x[n];
double y[n];

int main() {
    const double slope = 1.0;
    const double y_int = 0.5;

    // set to 8 threads (since 8 threads gives better parallelization)
    int num_threads = 8;
    omp_set_num_threads(num_threads);
        
    // Initialize the input vectors
    // All points lie on the line y = 1*x + 0.5
    #pragma omp parallel for
    for (int i = 0; i < n; i += 4) {
        x[i] = i;
        y[i] = slope * x[i] + y_int;
        x[i+1] = (i+1);
        y[i+1] = slope * x[i+1] + y_int;
        x[i+2] = (i+2);
        y[i+2] = slope * x[i+2] + y_int;
        x[i+3] = (i+3);
        y[i+3] = slope * x[i+3] + y_int;        
    }

    double m1 = 0, b1 = 0;
    bool rv1;

    // Use CUDA to calculate least squares
    rv1 = calcLeastSquaresCUDA(x, y, n, &m1, &b1);

    printf("slope = %6.4lf, m1 = %6.4lf\n", slope, m1);
    cout << "y_int = " << y_int << ", b1 = " << b1 << endl;
    cout << "rv1 = " << rv1 << endl;

    return 0;
}